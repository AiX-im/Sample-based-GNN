#include "hip/hip_runtime.h"
#include"ntsCudaTest.cuh"
#include <iostream>

__global__ void test_cuda(double* res, int* M, int* N)
{
    double sum = 0.0;
    for(int j =0; j< *M; j++)
    {
        for(int i = 0; i < *N; i++)
        {
          sum = sum + i *(*M);
        }
    }
    *res = sum;
    // std::printf("test sum: %lf\n", *res);
}
void testCuda(double* res, hipStream_t stream, int M, int N) {
    std::printf("进行了cuda调用\n");
    double* d_A;
    int *d_M;
    int *d_N;
    int nBytes = sizeof(double);
    int mBytes = sizeof(int);
    hipMallocAsync((double**)&d_A, nBytes, stream);
    hipMallocAsync((int**)&d_M, mBytes, stream);
    hipMallocAsync((int**)&d_N, mBytes, stream);
    hipMemcpyAsync(d_M, &M, mBytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_N, &N, mBytes, hipMemcpyHostToDevice, stream);
    // test_cuda<<<CUDA_NUM_BLOCKS,CUDA_NUM_THREADS,0,stream>>>(d_A, d_M, d_N);
    hipMemcpyAsync(res, d_A, nBytes, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    std::printf("res: %lf\n", *res);
}

void testStream(hipStream_t stream) {
    int num = 10 * 1024 * 1024 / sizeof(int);
    int * h_a = new int[num];
    int nBytes = num * sizeof(int);
    int * d_a;
    hipMallocAsync((int **)&d_a, nBytes, stream);
    hipMemcpyAsync(d_a, h_a, nBytes, hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);
    hipFreeAsync(d_a, stream);
}